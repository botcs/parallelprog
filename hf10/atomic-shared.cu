#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void first(float *input, float *output, int n)
{
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    

    __shared__ float tmp[1024];
    //exter
    tmp[threadIdx.x] = input[tid];
    if(threadIdx.x == 0){
        float sum = 0;
        for (int i = 0; i < blockDim.x; i++) sum += tmp[i];
        atomicAdd(&output[0], sum);
    }
    
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 1 << 28;
 
    //Host vector
    float *h_c;
 
    //Device output vector
    float *d_c;
    float *d_d;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);
 
    // Allocate memory on host
    h_c = (float*)malloc(bytes);
    for (int i = 0; i < n; i++)
        //h_c[i] = i;
        h_c[i] = (float)rand()/(float)(RAND_MAX);
 
    // Allocate memory on GPU
    cudaCheck(hipMalloc(&d_c, bytes));
    cudaCheck(hipMalloc(&d_d, sizeof(float)));
    cudaCheck(hipMemset(d_d, 0, sizeof(float)));
    cudaCheck(hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice));
 
    // Copy host vectors to device
    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    double t1 = omp_get_wtime();
    // Execute the kernel
    first<<<gridSize, blockSize>>>(d_c, d_d, n);
    // Synchronize
    cudaCheck(hipDeviceSynchronize());
    double elapsed = omp_get_wtime() - t1;      
    printf("Time: %g\n", elapsed);

    // Copy array back to host
    cudaCheck(hipMemcpy( h_c, d_d, sizeof(float), hipMemcpyDeviceToHost ));
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    printf("%g\n", h_c[0]);

 
    // Release device memory
    hipFree(d_c);
    hipFree(d_d);
 
    // Release host memory
    free(h_c);
 
    return 0;
}


